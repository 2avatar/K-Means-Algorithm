#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <mpi.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "Header.h"

__global__ void fillDistancePointFromPoint(float *distanceArray, Point *points, int numOfThreadsPerBlock, int numOfPoints, int iteration) {

	int index = threadIdx.x + (blockIdx.x * numOfThreadsPerBlock);

	if (index < numOfPoints) {

		float x = points[index].x - points[iteration].x;
		float y = points[index].y - points[iteration].y;

		distanceArray[index] = x*x + y*y;

	}
}

__global__ void fillPointIndexArray(Group *groups, Point *points, int *pointIndex, int numOfPoints, int numOfKlusters, int numOfThreadsPerBlock) {

	int index = threadIdx.x + (blockIdx.x * numOfThreadsPerBlock);
	float closestDistance, tempClosestDistance;
	float x, y;

	if (index < numOfPoints) {

		// assume first kluster to be closest
		pointIndex[index] = 0;
		x = groups[0].kluster.point.x - points[index].x;
		y = groups[0].kluster.point.y - points[index].y;

		closestDistance = x*x + y*y;

		// check other klusters to be closest
		for (int j = 0; j < numOfKlusters; j++) {

			x = groups[j].kluster.point.x - points[index].x;
			y = groups[j].kluster.point.y - points[index].y;

			tempClosestDistance = x*x + y*y;

			if (tempClosestDistance < closestDistance) {
				// new closest kluster and distance
				closestDistance = tempClosestDistance;
				pointIndex[index] = j;
			}
		}
	}

}

hipError_t findGroupDiameterWithCuda(Group *group) {

	hipError_t cudaStatus;
	float *dev_distanceArray;
	float maxDiameterArray;
	float *h_distanceArray;
	Point *dev_points;
	int numOfThreadsPerBlock = 500;
	int numOfPoints = group->numOfPoints;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	h_distanceArray = (float*)malloc(numOfPoints * sizeof(float));

	cudaStatus = hipMalloc((void**)&dev_distanceArray, numOfPoints * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_points, numOfPoints * sizeof(Point));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// copy data from host
	cudaStatus = hipMemcpy(dev_points, group->points, numOfPoints * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// + 1 in case of 3.2 = 3
	int numOfBlocks = ((int)(numOfPoints / numOfThreadsPerBlock) + 1);
	group->klusterDiameter = 0;
	for (int i = 0; i < numOfPoints; i++) {

		
		fillDistancePointFromPoint << <numOfBlocks, numOfThreadsPerBlock >> > (dev_distanceArray, dev_points, numOfThreadsPerBlock, numOfPoints, i);

		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(h_distanceArray, dev_distanceArray, numOfPoints * sizeof(float), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		for (int j = 0; j < numOfPoints; j++) {
			if (h_distanceArray[j] > group->klusterDiameter) {
				group->klusterDiameter = h_distanceArray[j];
			}
		}
	}

	group->klusterDiameter = sqrtf(group->klusterDiameter);
	//printf("%.3f \n", group->klusterDiameter);

Error:

	free(h_distanceArray);
	hipFree(dev_points);
	hipFree(dev_distanceArray);

	return cudaStatus;
}

hipError_t groupPointsToKlustersCudaHelper(Group *groups, Point *points, int *pointIndex, int numOfPoints, int numOfKlusters) {

	Group *dev_groups;
	Point *dev_points;
	int *dev_pointIndex;
	hipError_t cudaStatus;
	int numOfBlocks;
	int numOfThreadsPerBlock = 500;


	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_points, numOfPoints * sizeof(Point));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_pointIndex, numOfPoints * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_groups, numOfKlusters * sizeof(Group));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	// copy data from host
	cudaStatus = hipMemcpy(dev_groups, groups, numOfKlusters * sizeof(Group), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!1");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_points, points, numOfPoints * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!2");
		goto Error;
	}

	// + 1 in case of 3.2 = 3
	numOfBlocks = ((int)(numOfPoints / numOfThreadsPerBlock) + 1);

	fillPointIndexArray << <numOfBlocks, numOfThreadsPerBlock >> >(dev_groups, dev_points, dev_pointIndex, numOfPoints, numOfKlusters, numOfThreadsPerBlock);

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(pointIndex, dev_pointIndex, numOfPoints * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!3");
		goto Error;
	}

	// free allocations
Error:
	hipFree(dev_points);
	hipFree(dev_pointIndex);
	hipFree(dev_groups);

	return cudaStatus;

}